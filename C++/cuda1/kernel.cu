
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <inttypes.h>
#include <chrono>

#define M_PI 3.14159265359

#define N 1000

uint64_t timeSinceEpochMillisec() {
  using namespace std::chrono;
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

class Point {
    public:
        float x = 0;
        float y = 0;
        float z = 0;
};


float yaw = 45*(M_PI/180);
float pitch = 0*(M_PI/180);
float roll = 0*(M_PI/180);

float a = cos(yaw)*cos(pitch);
float b = cos(yaw)*sin(pitch)*sin(roll)-sin(yaw)*cos(roll);
float c = cos(yaw)*sin(pitch)*cos(roll)+sin(yaw)*sin(roll);
float d = sin(yaw)*cos(pitch);
float e = sin(yaw)*sin(pitch)*sin(roll)+cos(yaw)*cos(roll);
float f = sin(yaw)*sin(pitch)*cos(roll)-cos(yaw)*sin(roll);
float g = -sin(pitch);
float h = cos(pitch)*sin(roll);
float i = cos(pitch)*cos(roll);


__global__ void MatrixMultiply(Point *points, float *matrix, Point *newpoints, int numPoints) {

    int k = threadIdx.x + blockIdx.x * blockDim.x;

    if (k < numPoints) {
        int x = points[k].x;
        int y = points[k].y;
        int z = points[k].z;

        
        newpoints[k].x = x*matrix[0]+y*matrix[1]+z*matrix[2];
        newpoints[k].y = x*matrix[3]+y*matrix[4]+z*matrix[5];
        newpoints[k].z = x*matrix[6]+y*matrix[7]+z*matrix[8];
    }



}


int main() {


    // compute size of point array
    int numPoints = 10000000;
    size_t arraySize = numPoints * sizeof(Point);
    // allocate point arrays
    Point *h_points = (Point *)malloc(arraySize);
    Point *h_newpoints = (Point *)malloc(arraySize);


    // compute size of matrix
    size_t matrixSize = 9 * sizeof(float);
    // allocate matrix values
    float *h_matrix = (float *)malloc(matrixSize);

    // check allocations
    if (h_points == NULL || h_matrix == NULL || h_newpoints == NULL) {
        printf("Failed to allocate host vectors!\n");
        return 1;
    }

    // initialize points
    for (int i = 0; i < numPoints; i++) {
        h_points[i].x = i;
        h_points[i].y = i;
        h_points[i].z = i;
    }

    // initialize matrix
    h_matrix[0] = a;
    h_matrix[1] = b;
    h_matrix[2] = c;
    h_matrix[3] = d;
    h_matrix[4] = e;
    h_matrix[5] = f;
    h_matrix[6] = g;
    h_matrix[7] = h;
    h_matrix[8] = i;

    // allocate memory on device
    Point *d_points = NULL;
    hipMalloc((void **)&d_points, arraySize);
    Point *d_newpoints = NULL;
    hipMalloc((void **)&d_newpoints, arraySize);
    float *d_matrix = NULL;
    hipMalloc((void **)&d_matrix, matrixSize);

    // copy points and matrix onto device
    hipMemcpy(d_points, h_points, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);

    // calculate threads/block distribution
    int threadsPerBlock = 256;
    int blocks = arraySize + threadsPerBlock / threadsPerBlock;

    // run kernel
    MatrixMultiply<<<blocks, threadsPerBlock>>>(d_points, d_matrix, d_newpoints, arraySize);

    uint64_t zero = timeSinceEpochMillisec();

    // copy results from device to host
    hipMemcpy(h_newpoints, d_newpoints, arraySize, hipMemcpyDeviceToHost);

    printf("%" PRIu64 "ms\n", timeSinceEpochMillisec() - zero);

    /*for (int i = 0; i < 300; i++) {
        printf("%f : %f\n", h_points[i].x, round(h_newpoints[i].x));
    }*/

    // free device memory
    hipFree(d_points);
    hipFree(d_newpoints);
    hipFree(d_matrix);

    // free host memory
    free(h_points);
    free(h_newpoints);
    free(h_matrix);

    
    printf("Done\n");

    return 0;
}



/*



__global__ void MatrixMultiply(Point *p) {

    int k = threadIdx.x;

    int x = p[k].x;
    int y = p[k].y;
    int z = p[k].z;

    p[k].x = x*a+y*b+z*c;
    p[k].y = x*d+y*e+f*z;
    p[k].z = x*g+y*h+i*z;



for (int i = 0; i < N; i++) {
        points[i].x = 2;
        points[i].y = 2;
        points[i].z = 2; 
    }


}

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <inttypes.h>
#include <chrono>

#define M_PI 3.14159265359 // pi constant
#define DEGTORAD M_PI/180 // conversion factor from degrees to radians
#define N 1000 // number of points

// function used for stopwatch
uint64_t timeSinceEpochMillisec() {
  using namespace std::chrono;
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

class Point {
    public:
        float x = 0;
        float y = 0;
        float z = 0;
};

// rotation vectors
float yaw = 45 * DEGTORAD;
float pitch = 0 * DEGTORAD;
float roll = 0 * DEGTORAD;

// pre-calculate matrix constants - ultimately this would be run once a frame
float a = cos(yaw)*cos(pitch);
float b = cos(yaw)*sin(pitch)*sin(roll)-sin(yaw)*cos(roll);
float c = cos(yaw)*sin(pitch)*cos(roll)+sin(yaw)*sin(roll);
float d = sin(yaw)*cos(pitch);
float e = sin(yaw)*sin(pitch)*sin(roll)+cos(yaw)*cos(roll);
float f = sin(yaw)*sin(pitch)*cos(roll)-cos(yaw)*sin(roll);
float g = -sin(pitch);
float h = cos(pitch)*sin(roll);
float i = cos(pitch)*cos(roll);

// kernel to multiply by the matrix constants
__global__ void MatrixMultiply(Point *points, float *matrix, Point *newpoints, int numPoints) {
    // calculate index
    int k = threadIdx.x + blockIdx.x * blockDim.x;

    // controls that kernel will only work for number of points
    if (k < numPoints) {
        // retrieves vector from GPU memory
        int x = points[k].x;
        int y = points[k].y;
        int z = points[k].z;

        // multiply vector by matrix
        newpoints[k].x = x*matrix[0]+y*matrix[1]+z*matrix[2];
        newpoints[k].y = x*matrix[3]+y*matrix[4]+z*matrix[5];
        newpoints[k].z = x*matrix[6]+y*matrix[7]+z*matrix[8];
    }
}

int main() {

    size_t arraySize = N * sizeof(Point); // compute size of point array

    // allocate point arrays
    Point *h_points = (Point *)malloc(arraySize);
    Point *h_newpoints = (Point *)malloc(arraySize);

    size_t matrixSize = 9 * sizeof(float); // compute size of matrix

    float *h_matrix = (float *)malloc(matrixSize); // allocate matrix values

    // check allocations
    if (h_points == NULL || h_matrix == NULL || h_newpoints == NULL) {
        printf("Failed to allocate vectors on host\n");
        return 1;
    }

    // initialize points
    for (int i = 0; i < N; i++) {
        h_points[i].x = i;
        h_points[i].y = i;
        h_points[i].z = i;
    }

    // initialize matrix
    h_matrix[0] = a; h_matrix[1] = b; h_matrix[2] = c;
    h_matrix[3] = d; h_matrix[4] = e; h_matrix[5] = f;
    h_matrix[6] = g; h_matrix[7] = h; h_matrix[8] = i;

    // allocate memory on device
    Point *d_points = NULL;
    hipMalloc((void **)&d_points, arraySize);
    Point *d_newpoints = NULL;
    hipMalloc((void **)&d_newpoints, arraySize);
    float *d_matrix = NULL;
    hipMalloc((void **)&d_matrix, matrixSize);

    // copy points and matrix onto device
    hipMemcpy(d_points, h_points, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);

    // calculate threads/block distribution
    int threadsPerBlock = 256;
    int blocks = arraySize + threadsPerBlock / threadsPerBlock;

    // run kernel
    MatrixMultiply<<<blocks, threadsPerBlock>>>(d_points, d_matrix, d_newpoints, arraySize);

    // copy results from device to host
    hipMemcpy(h_newpoints, d_newpoints, arraySize, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_points);
    hipFree(d_newpoints);
    hipFree(d_matrix);

    // free host memory
    free(h_points);
    free(h_newpoints);
    free(h_matrix);

    
    printf("Done\n");

    return 0;
}
